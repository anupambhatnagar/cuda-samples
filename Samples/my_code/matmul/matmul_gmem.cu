#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vector>

int div_ceil(int a, int b) { return (a % b != 0) ? (a / b + 1) : a / b; }

template <int block_size>
__global__ void matmul(const int *da, const int *db, int *dc, int n) {
  // compute each entry of the matrix using row and col
  // we need the row from A matrix and column from B matrix.
  // sum over the rolling index

  int row = blockIdx.x * block_size + (threadIdx.x / block_size);
  int col = blockIdx.y * block_size + (threadIdx.x % block_size);

  if (col < n && row < n) {
    dc[row * n + col] = 0;
    for (int i = 0; i < n; i++) {
      dc[row * n + col] += da[row * n + i] * db[i * n + col];
    }
  }
}

void verify_result(std::vector<int> a, std::vector<int> b, std::vector<int> c,
                   int n) {
  for (int row = 0; row < n; row++) {
    for (int col = 0; col < n; col++) {
      int tmp = 0;
      for (int k = 0; k < n; k++) {
        tmp += a[row * n + k] * b[k * n + col];
      }
      assert(tmp == c[row * n + col]);
    }
  }
}

int main() {
  const int n = 1<<10;
  size_t bytes = sizeof(int) * n * n;

  // allocate memory on host
  std::vector<int> ha(n * n);
  std::vector<int> hb(n * n);
  std::vector<int> hc(n * n);

  generate(ha.begin(), ha.end(), []() { return 1; });
  generate(hb.begin(), hb.end(), []() { return 2; });

  // allocate memory on device
  int *da, *db, *dc;

  checkCudaErrors(hipMalloc(&da, bytes));
  checkCudaErrors(hipMalloc(&db, bytes));
  checkCudaErrors(hipMalloc(&dc, bytes));

  // copy data from host to device
  checkCudaErrors(hipMemcpy(da, ha.data(), bytes, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(db, hb.data(), bytes, hipMemcpyHostToDevice));

  // launch kernel
  const uint threads_per_block = 32;
  const uint block_size = 16;

  dim3 blocks(threads_per_block * threads_per_block);
  dim3 grids(div_ceil(n, blocks.x), div_ceil(n, blocks.y));

  matmul<block_size><<<grids, blocks>>>(da, db, dc, n);

  // copy result from device to host
  checkCudaErrors(hipMemcpy(hc.data(), dc, bytes, hipMemcpyDeviceToHost));

  // unit test
//  for (int i=0; i < hc.size(); i++){
//    std::cout<<"hc[i] = " << hc[i] << "\n";
//  }
//  verify_result(ha, hb, hc, n);

  // free memory on device
  // vector destructor frees the memory on host when vectors go out of scope
  checkCudaErrors(hipFree(da));
  checkCudaErrors(hipFree(db));
  checkCudaErrors(hipFree(dc));

  return 0;
}
