#include "hip/hip_runtime.h"
// System includes
#include <assert.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

/* Add two vectors on the GPU */
__global__ void vectorAddGPU(float *a, float *b, float *c, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}

// Allocate generic memory with malloc() and pin it later instead of using
// hipHostAlloc()
bool bPinGenericMemory = true;

// Macro to aligned up to the memory size in question
#define MEMORY_ALIGNMENT 4096
#define ALIGN_UP(x, size) (((size_t)x + (size - 1)) & (~(size - 1)))

int main(int argc, char **argv) {
  int n, nelem;
  size_t bytes;
  float *a, *b, *c;           // Pinned memory allocated on the CPU
  float *a_UA, *b_UA, *c_UA;  // Non-4K Aligned Pinned memory on the CPU
  float *d_a, *d_b, *d_c;     // Device pointers for mapped memory
  float errorNorm, refNorm, ref, diff;

  /* Allocate mapped CPU memory. */
  nelem = 1048576;
  bytes = nelem * sizeof(float);

  a_UA = (float *)malloc(bytes + MEMORY_ALIGNMENT);
  b_UA = (float *)malloc(bytes + MEMORY_ALIGNMENT);
  c_UA = (float *)malloc(bytes + MEMORY_ALIGNMENT);

  // We need to ensure memory is aligned to 4K (so we will need to padd memory
  // accordingly)
  a = (float *)ALIGN_UP(a_UA, MEMORY_ALIGNMENT);
  b = (float *)ALIGN_UP(b_UA, MEMORY_ALIGNMENT);
  c = (float *)ALIGN_UP(c_UA, MEMORY_ALIGNMENT);

  checkCudaErrors(hipHostRegister(a, bytes, hipHostRegisterMapped));
  checkCudaErrors(hipHostRegister(b, bytes, hipHostRegisterMapped));
  checkCudaErrors(hipHostRegister(c, bytes, hipHostRegisterMapped));

  /* Initialize the vectors. */
  for (n = 0; n < nelem; n++) {
    a[n] = rand() / (float)RAND_MAX;
    b[n] = rand() / (float)RAND_MAX;
  }

  /* Get the device pointers for the pinned CPU memory mapped into the GPU
       memory space. */
  #if CUDART_VERSION >= 2020
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_a, a, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)d_b, b, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_c, c, 0));
  #endif

  /* Call the GPU kernel using the CPU pointers residing in CPU mapped memory.
   */
  printf("> vectorAddGPU kernel will add vectors using mapped CPU memory...\n");

  dim3 block(256);
  dim3 grid((unsigned int)ceil(nelem / (float)block.x));
  vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, nelem);

  checkCudaErrors(hipDeviceSynchronize());
  getLastCudaError("vectorAddGPU() execution failed");

  /* Compare the results */
  printf("> Checking the results from vectorAddGPU() ...\n");
  errorNorm = 0.f;
  refNorm = 0.f;

  for (n = 0; n < nelem; n++) {
    ref = a[n] + b[n];
    diff = c[n] - ref;
    errorNorm += diff * diff;
    refNorm += ref * ref;
  }

  errorNorm = (float)sqrt((double)errorNorm);
  refNorm = (float)sqrt((double)refNorm);
  printf("errorNorm is %f\n", errorNorm);
  printf("refNorm is %f\n", refNorm);

  /* Memory clean up */
  printf("> Releasing CPU memory...\n");

  checkCudaErrors(hipHostUnregister(a));
  checkCudaErrors(hipHostUnregister(b));
  checkCudaErrors(hipHostUnregister(c));
  free(a_UA);
  free(b_UA);
  free(c_UA);

  exit(errorNorm / refNorm < 1.e-6f ? EXIT_SUCCESS : EXIT_FAILURE);
}
